#include "hip/hip_runtime.h"
/***************************************************************************
 *                             KNN-Cpp                                     *
 *                                                                         *
 * Software to analyze processing times to Knn networks using Iris dataset *
 *                                                                         *
 *             Serial processing: using CPU and one thread;                *
 *         Parallel processing: using CPU and multiple threads;            *
 *           Cuda processing: using GPU and multiple threads.              *
 *                                                                         *
 *    User can select the number of tests, K and select desire type of     *
 *          calculation. Results are shown as text and graphics.           *
 *                                                                         *
 * ======================================================================= *
 *                      Autor: Rodrigo Aguiar                              *
 *           rodrigo.b.aguiar@gmail.com / rbaguiar@id.uff.br               *
 ***************************************************************************/
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#include <thrust/device_vector.h>

#include <stdio.h>
#include "knnCuda.h"
#include <string>


//struct KNNData;
struct KNNData{
    double sepalLenght = 0.0;
    double sepalWidth = 0.0;
    double petalLenght = 0.0;
    double petalWidht = 0.0;
    std::string irisClass = "";
    int id = 0;
};

__global__ void classify(KNNData *trainningCases, KNNData *testCase, int *K){


    printf("Hello from GPU! \n");



//    auto setosa = 0;
//    auto versicolor = 0;
//    auto virginica = 0;
//    int neighborCounter = 0;
//    std::set<std::pair<double, int> >::iterator it;
//    std::set<std::pair<double, int> > distances;

//    std::cout << std::this_thread::get_id() << "\n";

//    for(int i = 0; i < cases; i++){
//        distances.insert(std::make_pair(euclidianDistance(trainningCases[i], testCase), i));
//    }

//    for (auto it = distances.begin(); it != distances.end(); it++){
//        if(neighborCounter == K)
//            break;

//        std::string classe = trainningCases[it->second].irisClass;

//        if(classe == "Iris-setosa")
//            setosa++;
//        else if(classe == "Iris-versicolor")
//            versicolor++;
//        else if(classe == "Iris-virginica")
//            virginica++;

//        neighborCounter++;
//    }

//    std::mutex mtx;
//    mtx.lock();
//    classFound[caseIndex] = returnClassification(setosa, versicolor, virginica);
//    mtx.unlock();

}

/**
 * Init CUDA.
 * Receive required data, sende data to device and acces functions.
 * @param trainningCases Array with trainning data.
 * @param testCase Specific entry to analyse
 * @param nCases Number of trainned cases
 * @param nTests Number of test cases
 * @param K Number of neighbors to analyse
*/
void initCuda(KNNData *&trainningCases, KNNData *&testCase, int *nCases, int *nTests, int* K){

    //receive data
    KNNData *deviceTrainningCases;
    KNNData *deviceTestCases;
    int *deviceK;

    int sizeTrainning = (sizeof(KNNData)**nCases);
    int sizeTest = (sizeof(KNNData)**nTests);

    if(hipSuccess != hipMalloc((void **)&deviceTrainningCases, sizeTrainning)) printf("Error allocating trainning cases. \n");
    if(hipSuccess != hipMalloc((void **)&deviceTestCases, sizeTest)) printf("Error allocating test cases. \n");
    if(hipSuccess != hipMalloc((void **)&deviceK, sizeof(int))) printf("Error allocating K. \n");

    if(hipSuccess != hipMemcpy(deviceTrainningCases, trainningCases, sizeTrainning, hipMemcpyHostToDevice)) printf("Error copying trainning cases. \n");
    if(hipSuccess != hipMemcpy(deviceTestCases, testCase, sizeTest, hipMemcpyHostToDevice)) printf("Error copying test cases. \n");
    if(hipSuccess != hipMemcpy(deviceK, K, sizeof(int), hipMemcpyHostToDevice)) printf("Error copying K. \n");


    printf("Sepal Widht: %4.2f \n", deviceTrainningCases[0].sepalWidth);
    printf("Petal Widht: %4.2f \n", deviceTestCases[0].petalWidht);
    printf("K: %d \n", deviceK);


    classify<<<1,1>>>(deviceTrainningCases, deviceTestCases, deviceK);


    //copiar resultados de volta para o host



    hipFree(deviceTrainningCases);
    hipFree(deviceTestCases);
    hipFree(deviceK);
    //clean

}


//int h_arr[N] = {1,2,3,4,5,6,7,8,9,10};
//StructA h_a;
//int *d_arr;

// 1. Allocate device array.
//hipMalloc((void**) &(d_arr), sizeof(int)*N);

// 2. Copy array contents from host to device.
//hipMemcpy(d_arr, h_arr, sizeof(int)*N, hipMemcpyHostToDevice);

// 3. Point to device pointer in host struct.
//h_a.arr = d_arr;

// 4. Call kernel with host struct as argument
//kernel2<<<N,1>>>(h_a);

// 5. Copy pointer from device to host.
//hipMemcpy(h_arr, d_arr, sizeof(int)*N, hipMemcpyDeviceToHost);

// 6. Point to host pointer in host struct
//    (or do something else with it if this is not needed)
//h_a.arr = h_arr;


